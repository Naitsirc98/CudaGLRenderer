#include "hip/hip_runtime.h"
﻿#pragma once

#include "engine/graphics/postfx/InversionPostFX.cuh"
#include "engine/graphics/postfx/CUDACommons.h"
#include <math.h>

namespace utad
{
	__global__ void kernel_Inversion(Pixel* pixels, int width, int height)
	{
		const int x = CUDA_X_POS;
		const int y = CUDA_Y_POS;
		if (x >= width || y >= height) return;

		Pixel& pixel = pixels[CUDA_INDEX_XY(x, y, width)];

		pixel.r = 255 - pixel.r;
		pixel.g = 255 - pixel.g;
		pixel.b = 255 - pixel.b;
	}

	void executeInversionFX(const RenderInfo& info)
	{
		dim3 gridSize;
		dim3 blockSize;
		Cuda::getKernelDimensions(gridSize, blockSize, info.width, info.height);

		Pixel* pixels = (Pixel*)info.d_pixels;

		kernel_Inversion<<<gridSize, blockSize>>>(pixels, info.width, info.height);
	}
}