﻿#include "engine/graphics/postfx/GaussianBlurFX.cuh"
#include <math.h>

namespace utad
{
    GaussianBlurFX::GaussianBlurFX() 
        : ConvolutionFilterFX(createFilter(), 5)
    {
    }

    const float* GaussianBlurFX::createFilter()
    {
        float* h_filter = new float[25] {
                1, 1, 1, 1, 1,
                1, 2, 2, 2, 1,
                1, 2, 3, 2, 1,
                1, 2, 2, 2, 2,
                1, 1, 1, 1, 1
        };

        for (int i = 0; i < 25; ++i) h_filter[i] /= 35.0f;

        return h_filter;
    }
}