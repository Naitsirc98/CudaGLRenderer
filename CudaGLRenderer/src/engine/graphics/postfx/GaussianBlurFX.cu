#include "hip/hip_runtime.h"
﻿#include "engine/graphics/postfx/GaussianBlurFX.cuh"
#include <math.h>

namespace utad
{
    __device__ float clamp(float value, float min, float max)
    {
        if (value < min) return min;
        if (value > max) return max;
        return value;
    }

	__global__ void kernel_GaussianBlur(CudaSurface colorBuffer, int width, int height,
        float* filter, int filterWidth, int filterHalfWidth)
	{
        const int x = CUDA_X_POS;
        const int y = CUDA_Y_POS;
        if (x >= width || y >= height) return;

        Pixel pixel;
        surf2Dread(&pixel, colorBuffer, x * 4, y);

        float r = 0.0f;
        float g = 0.0f;
        float b = 0.0f;
        float a = 0.0f;

        for (int i = -filterHalfWidth; i <= filterHalfWidth; ++i)
        {
            for (int j = -filterHalfWidth; j <= filterHalfWidth; ++j)
            {
                int row = clamp(y + i, 0, height - 1);
                int column = clamp(x + j, 0, width - 1);

                Pixel p;// = pixels[CUDA_INDEX_XY(column, row, width)];
                surf2Dread(&p, colorBuffer, column * 4, row);
                float f = filter[CUDA_INDEX_XY((j + filterHalfWidth), (i + filterHalfWidth), filterWidth)];

                r += p.x * f;
                g += p.y * f;
                b += p.z * f;
                a += p.w * f;
            }
        }

        pixel.x = r;
        pixel.y = g;
        pixel.z = b;
        pixel.w = a;

        surf2Dwrite(pixel, colorBuffer, x * 4, y);
	}


    GaussianBlurFX::GaussianBlurFX(size_t filterWidth) 
        : m_FilterWidth(filterWidth), m_FilterHalfWidth(filterWidth/2)
    {
        initializeFilter();
    }

    GaussianBlurFX::~GaussianBlurFX()
    {
        Cuda::free(m_D_GaussianBlurFilter);
        m_D_GaussianBlurFilter = nullptr;
    }

    void GaussianBlurFX::execute(const PostFXInfo& info)
	{
        dim3 gridSize;
        dim3 blockSize;
        Cuda::getKernelDimensions(gridSize, blockSize, info.width, info.height);

		kernel_GaussianBlur<<<gridSize, blockSize>>>(
            info.colorBuffer, 
            info.width, 
            info.height,
            m_D_GaussianBlurFilter,
            m_FilterWidth,
            m_FilterHalfWidth
            );	
    }

    void GaussianBlurFX::initializeFilter()
    {
        const float sigma = 2.0f;
        const int w = m_FilterWidth;
        const int wh = m_FilterHalfWidth;
        const int size = w * w;

        float* h_filter = new float[size];

        float sum = 0.0f;

        for (int r = -w / 2; r <= w / 2; ++r) {
            for (int c = -w / 2; c <= w / 2; ++c) {
                float f = expf(-(float)(c * c + r * r) / (2.f * sigma * sigma));
                h_filter[(r + w / 2) * w + c + w / 2] = f;
                sum += f;
            }
        }

        float n = 1.0f / sum;

        for (int r = -w / 2; r <= w / 2; ++r) {
            for (int c = -w / 2; c <= w / 2; ++c) {
                h_filter[(r + w / 2) * w + c + w / 2] *= n;
            }
        }

        m_D_GaussianBlurFilter = (float*)Cuda::malloc(size * sizeof(float));
        Cuda::copyHostToDevice(h_filter, m_D_GaussianBlurFilter, size * sizeof(float));

        delete[] h_filter;
    }
}