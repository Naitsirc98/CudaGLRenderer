#include "hip/hip_runtime.h"
﻿#include "engine/graphics/postfx/GammaCorrectionFX.cuh"
#include <math.h>

namespace utad
{
	__global__ void kernel_GammaCorrection(CudaSurface colorBuffer, int width, int height, float exposure)
	{
		static const float gamma = 1.0f / 2.2f;

		const int x = CUDA_X_POS;
		const int y = CUDA_Y_POS;
		if (x >= width || y >= height) return;

		Pixel pixel;
		surf2Dread(&pixel, colorBuffer, x * 4, y);

		float r = pixel.x / 255.0f;
		float g = pixel.y / 255.0f;
		float b = pixel.z / 255.0f;
		float a = pixel.w / 255.0f;

		// Tone Mapping
		r = 1.0f - exp(-r * exposure);
		g = 1.0f - exp(-g * exposure);
		b = 1.0f - exp(-b * exposure);
		a = 1.0f - exp(-a * exposure);

		// Gamma Correction
		r = powf(r, gamma);
		g = powf(g, gamma);
		b = powf(b, gamma);
		a = powf(a, gamma);

		pixel.x = (unsigned char)(r * 255.0f);
		pixel.y = (unsigned char)(g * 255.0f);
		pixel.z = (unsigned char)(b * 255.0f);
		pixel.w = (unsigned char)(a * 255.0f);
	
		surf2Dwrite(pixel, colorBuffer, x * 4, y);
	}

	void GammaCorrectionFX::execute(const PostFXInfo& info)
	{
		dim3 gridSize;
		dim3 blockSize;
		Cuda::getKernelDimensions(gridSize, blockSize, info.width, info.height);

		kernel_GammaCorrection<<<gridSize, blockSize>>>(info.colorBuffer, info.width, info.height, info.exposure);
	}
}